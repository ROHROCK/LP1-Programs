
#include <hip/hip_runtime.h>
#include<iostream>
#include<time.h>
#define SIZE 100
using namespace std;

__global__ void mul(int (*mat1)[SIZE][SIZE] , int (*mat2)[SIZE][SIZE] , long (*result)[SIZE][SIZE]){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j =  threadIdx.y + blockIdx.y * blockDim.y;
    if(i < SIZE && j < SIZE){
        (*result)[i][j] = 0;
        for(int k = 0 ; k < SIZE ; k++)
            (*result)[i][j] += (*mat1)[i][k]*(*mat2)[k][j];
    }
}

int main(){
    int mat1[SIZE][SIZE];
    int mat2[SIZE][SIZE];
    long result[SIZE][SIZE];

    // pointer to gpu location
    int (*d_in_mat1)[SIZE][SIZE], (*d_in_mat2)[SIZE][SIZE];
    long (*d_out_result)[SIZE][SIZE];

    // intialize
    for(int i = 0 ; i < SIZE ; i++){
        for(int j = 0 ; j < SIZE ; j++){
            mat1[i][j] = i+1;
            mat2[i][j] = i+1;
            result[i][j] = 0;
        }
    }

    // Allocate memory to gpu
    hipMalloc((void**)&d_in_mat1,SIZE*SIZE*sizeof(int));
    hipMalloc((void**)&d_in_mat2,SIZE*SIZE*sizeof(int));
    hipMalloc((void**)&d_out_result,SIZE*SIZE*sizeof(long));

    // Copy the contents to gpu
    hipMemcpy(d_in_mat1,mat1,SIZE*SIZE*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_in_mat2,mat2,SIZE*SIZE*sizeof(int),hipMemcpyHostToDevice);

    // invoke the kernel function
    
    dim3 threadsPerBlock(SIZE, SIZE);
    dim3 blocksPerGrid(1, 1);
    
    if(SIZE*SIZE > 1024){
        threadsPerBlock.x = 1024;
        threadsPerBlock.y = 1024;
        blocksPerGrid.x = ceil(double(SIZE)/double(threadsPerBlock.x));
        blocksPerGrid.y = ceil(double(SIZE)/double(threadsPerBlock.y));
    }

    clock_t startTime = clock();
    // mul<<<blocksPerGrid,threadsPerBlock>>>(d_in_mat1,d_in_mat2,d_out_result);
    mul<<<blocksPerGrid, threadsPerBlock>>>(d_in_mat1,d_in_mat2,d_out_result);
    clock_t endTime = clock();
    printf("\n\nTime for GPU: %f",(float)(endTime-startTime)/CLOCKS_PER_SEC);

    // cpy the result back 
    hipMemcpy(result,d_out_result,SIZE*SIZE*sizeof(long),hipMemcpyDeviceToHost);
    printf("\nres GPU: %ld", result[0][0]);

    // sequential code
    startTime = clock();
    for(int i = 0 ; i < SIZE ; i++){
        for(int j = 0 ; j < SIZE ; j++){
            result[i][j] = 0;
            for(int k = 0 ; k < SIZE ; k++)
               result[i][j] += mat1[i][k]*mat2[k][j];
        }
    }
    printf("\nres seq: %ld", result[0][0]);
    endTime = clock();
    printf("\n\nTime for sequential: %f",(float)(endTime-startTime)/CLOCKS_PER_SEC);
    // print result
    // for(int i = 0 ; i < SIZE ; i++){
    //     cout<<result[i]<<" ";
    // }
    
    return 0;
}