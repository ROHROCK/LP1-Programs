
#include <hip/hip_runtime.h>
#include<iostream>
#include<time.h>
#define SIZE 100000
using namespace std;

__global__ void addVect(int *vect1 ,int *vect2 , int *resultVect){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    // printf("Thread id == %d || Block Id == %d\n",threadIdx.x,blockDim.x);
    resultVect[i] = vect1[i] + vect2[i];
}

int main(){
    int *d_inVect1,*d_inVect2,*d_outResultVector;
    int vect1[SIZE],vect2[SIZE];
    int resultVect[SIZE];
    hipEvent_t gpu_start,gpu_stop;
    float gpu_elapsed_time;
    // Initializing both the vectors
    for(int i = 0 ; i < SIZE ; i++){
        vect1[i] = i;
        vect2[i] = i;
    }
    // Parallel code

    // Allocate memory on GPU for 3 vectors
    hipMalloc((void**)&d_inVect1,SIZE*(sizeof(int)));
    hipMalloc((void**)&d_inVect2,SIZE*(sizeof(int)));
    hipMalloc((void**)&d_outResultVector,SIZE*(sizeof(int)));

    // CPY the vector contents
    hipMemcpy(d_inVect1,vect1,SIZE*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_inVect2,vect2,SIZE*sizeof(int),hipMemcpyHostToDevice);

    int blk = SIZE/1024;
    // Call the kernel
    clock_t startTime = clock();
    addVect<<<blk+1,1024>>>(d_inVect1,d_inVect2,d_outResultVector);
    clock_t endTime = clock();
    printf("\n\nTime for sequential: %.4f",(float)(endTime-startTime)/CLOCKS_PER_SEC);
    // cudaDeviceSynchronize();

    // Copy gpu mem to cpu mem
    hipMemcpy(resultVect,d_outResultVector,SIZE*sizeof(int),hipMemcpyDeviceToHost);


    cout<<"The time taken by GPU is :"<<gpu_elapsed_time<<endl;
   
    // for(int i = 0 ; i < SIZE ; i++){
    //     cout<<resultVect[i]<<" ";
    // }

    // Sequential code
    startTime = clock();
    int resultVect2[SIZE];
    for(int i = 0 ; i < SIZE ; i++){
        resultVect2[i] = vect1[i] * vect2[i];
    }
    // for(int i = 0 ; i < SIZE ; i++){
    //     cout<<resultVect2[i]<<" ";
    // }
    endTime = clock();
    printf("\n\nTime for sequential: %.4f",(float)(endTime-startTime)/CLOCKS_PER_SEC);

    return 0;
}