
#include <hip/hip_runtime.h>
#include<iostream>
#include<time.h>
#define SIZE 3
using namespace std;

__global__ void mul(int *vect , int (*mat)[SIZE][SIZE] , int *res){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    res[i] = 0;
    for(int j = 0 ; j < SIZE ; j++)
        res[i] += vect[j]* (*mat)[j][i];  
}

int main(){
    int vect[SIZE];
    int mat[SIZE][SIZE];
    int result[SIZE];
    hipEvent_t gpu_start,gpu_stop;
    float gpu_elapsed_time;

    // pointer to gpu location
    int *d_in_vector,(*d_in_mat)[SIZE][SIZE];
    int *d_out_result;

    // intialize
    for(int i = 0 ; i < SIZE ; i++){
        vect[i] = i;
        for(int j = 0 ; j < SIZE ; j++){
            mat[i][j] = i;
        }
    }

    // Allocate memory to gpu
    hipMalloc((void**)&d_in_vector,SIZE*sizeof(int));
    hipMalloc((void**)&d_in_mat,SIZE*SIZE*sizeof(int));
    hipMalloc((void**)&d_out_result,SIZE*sizeof(int));

    // Copy the contents to gpu
    hipMemcpy(d_in_vector,vect,SIZE*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_in_mat,mat,SIZE*SIZE*sizeof(int),hipMemcpyHostToDevice);

    // Start record for gpu_start
    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_stop);
    hipEventRecord(gpu_start,0);

    // invoke the kernel function
    mul<<<1,SIZE>>>(d_in_vector,d_in_mat,d_out_result);
    // cpy the result back 
    hipMemcpy(result,d_out_result,SIZE*sizeof(int),hipMemcpyDeviceToHost);
    
    hipEventRecord(gpu_stop,0);
    hipEventSynchronize(gpu_stop);
    hipEventElapsedTime(&gpu_elapsed_time,gpu_start,gpu_stop);
    hipEventDestroy(gpu_start);
    hipEventDestroy(gpu_stop);

    cout<<"The time taken by GPU is :"<<gpu_elapsed_time<<endl;
    
    
    // sequential code
    clock_t startTime = clock();
    for(int i = 0 ; i < SIZE ; i++){
        result[i] = 0;
        for(int j = 0 ; j < SIZE ; j++)
            result[i] += vect[j]*mat[j][i];  
    }
    clock_t endTime = clock();
    printf("\n\nTime for sequential: %.4f",(float)(endTime-startTime)/CLOCKS_PER_SEC);
    // print result
    // for(int i = 0 ; i < SIZE ; i++){
    //     cout<<result[i]<<" ";
    // }
    
    return 0;
}