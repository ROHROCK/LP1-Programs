
#include <hip/hip_runtime.h>
#include<iostream>
#include<time.h>
#define SIZE 1000
using namespace std;

__global__ void mul(int *vect , int (*mat)[SIZE][SIZE] , long *res){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    res[i] = 0;
    for(int j = 0 ; j < SIZE ; j++)
        res[i] += vect[j]* (*mat)[j][i];  
}

int main(){
    int vect[SIZE];
    int mat[SIZE][SIZE];
    long result[SIZE];
    hipEvent_t gpu_start,gpu_stop;
    float gpu_elapsed_time;

    // pointer to gpu location
    int *d_in_vector,(*d_in_mat)[SIZE][SIZE];
    long *d_out_result;

    // intialize
    for(int i = 0 ; i < SIZE ; i++){
        vect[i] = i;
        for(int j = 0 ; j < SIZE ; j++){
            mat[i][j] = i;
        }
    }

    // Allocate memory to gpu
    hipMalloc((void**)&d_in_vector,SIZE*sizeof(int));
    hipMalloc((void**)&d_in_mat,SIZE*SIZE*sizeof(int));
    hipMalloc((void**)&d_out_result,SIZE*sizeof(long));

    // Copy the contents to gpu
    hipMemcpy(d_in_vector,vect,SIZE*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_in_mat,mat,SIZE*SIZE*sizeof(int),hipMemcpyHostToDevice);

    // invoke the kernel function
    int blk = SIZE/1024;

    clock_t startTime = clock();
    mul<<<blk+1,1024>>>(d_in_vector,d_in_mat,d_out_result);
    clock_t endTime = clock();
    printf("\n\nTime for GPU: %f",(float)(endTime-startTime)/CLOCKS_PER_SEC);

    // cpy the result back 
    hipMemcpy(result,d_out_result,SIZE*sizeof(long),hipMemcpyDeviceToHost);
       
    // sequential code
    startTime = clock();
    for(int i = 0 ; i < SIZE ; i++){
        result[i] = 0;
        for(int j = 0 ; j < SIZE ; j++)
            result[i] += vect[j]*mat[j][i];  
    }
    endTime = clock();
    printf("\n\nTime for sequential: %f",(float)(endTime-startTime)/CLOCKS_PER_SEC);
    // print result
    // for(int i = 0 ; i < SIZE ; i++){
    //     cout<<result[i]<<" ";
    // }
    
    return 0;
}