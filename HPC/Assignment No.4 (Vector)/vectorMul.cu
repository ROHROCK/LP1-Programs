
#include <hip/hip_runtime.h>
#include<iostream>
#include<time.h>
#define SIZE 1000
using namespace std;

__global__ void mul(int *vect , int (*mat)[SIZE][SIZE] , long *res){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    res[i] = 0;
    for(int j = 0 ; j < SIZE ; j++)
        res[i] += vect[j]* (*mat)[j][i];  
}

int main(){
    int vect[SIZE];
    int mat[SIZE][SIZE];
    long result[SIZE];
    hipEvent_t gpu_start,gpu_stop;
    float gpu_elapsed_time;

    // pointer to gpu location
    int *d_in_vector,(*d_in_mat)[SIZE][SIZE];
    long *d_out_result;

    // intialize
    for(int i = 0 ; i < SIZE ; i++){
        vect[i] = i;
        for(int j = 0 ; j < SIZE ; j++){
            mat[i][j] = i;
        }
    }

    // Allocate memory to gpu
    hipMalloc((void**)&d_in_vector,SIZE*sizeof(int));
    hipMalloc((void**)&d_in_mat,SIZE*SIZE*sizeof(int));
    hipMalloc((void**)&d_out_result,SIZE*sizeof(long));

    // Copy the contents to gpu
    hipMemcpy(d_in_vector,vect,SIZE*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_in_mat,mat,SIZE*SIZE*sizeof(int),hipMemcpyHostToDevice);

    // Start record for gpu_start
    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_stop);
    hipEventRecord(gpu_start,0);

    // invoke the kernel function
    int blk = SIZE/1024;
    mul<<<blk+1,1024>>>(d_in_vector,d_in_mat,d_out_result);
    // cpy the result back 
    hipMemcpy(result,d_out_result,SIZE*sizeof(long),hipMemcpyDeviceToHost);
    
    hipEventRecord(gpu_stop,0);
    hipEventSynchronize(gpu_stop);
    hipEventElapsedTime(&gpu_elapsed_time,gpu_start,gpu_stop);
    hipEventDestroy(gpu_start);
    hipEventDestroy(gpu_stop);

    cout<<"The time taken by GPU is :"<<gpu_elapsed_time<<endl;
    
    
    // sequential code
    clock_t startTime = clock();
    for(int i = 0 ; i < SIZE ; i++){
        result[i] = 0;
        for(int j = 0 ; j < SIZE ; j++)
            result[i] += vect[j]*mat[j][i];  
    }
    clock_t endTime = clock();
    printf("\n\nTime for sequential: %.3f",(float)(endTime-startTime)/CLOCKS_PER_SEC);
    // print result
    // for(int i = 0 ; i < SIZE ; i++){
    //     cout<<result[i]<<" ";
    // }
    
    return 0;
}