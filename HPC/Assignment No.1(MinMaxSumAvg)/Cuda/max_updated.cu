
#include <hip/hip_runtime.h>
#include<stdio.h>
#define SIZE 1000000
// MAX_THREADS depends on type of GPU
#define MAX_THREADS 1024

__global__ void max(int* input, int* maxOut) {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        atomicMax(maxOut, input[i]);
}

int main() {
        int input[SIZE];
        int maxO = 0;
        int i = 0;
        for(i = 0; i < SIZE; i++)
                input[i] = (rand() % 10000) + 1;
        
        int* d_input;
        int* d_max;
        
        hipMalloc((void**)&d_input, SIZE * sizeof(int));
        hipMalloc((void**)&d_max, sizeof(int));
        
        hipMemcpy(d_input, &input, SIZE * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_max, &maxO, sizeof(int), hipMemcpyHostToDevice);
        // adjusting the number of threads per block
        dim3 threadsPerBlock(SIZE);
        dim3 blocksPerGrid(1, 1);
        if(SIZE > 1024){
                threadsPerBlock.x = 1024;
                blocksPerGrid.x = ceil(double(SIZE)/double(threadsPerBlock.x));
        }

        max<<<blocksPerGrid,threadsPerBlock>>>(d_input, d_max);
        
        hipMemcpy(&maxO, d_max, sizeof(int), hipMemcpyDeviceToHost);
        
        printf("\nMax: %d",maxO);
        
        hipFree(d_max);
        hipFree(d_input);  
        return 0;
}
