
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
#define SIZE 10000
// MAX_THREADS depends on type of GPU
#define MAX_THREADS 1024

__global__ void sum(const int* __restrict__ input, const int size, int* sumOut)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    atomicAdd(sumOut, input[i]);
}

int main()
{
  int i;
  int a[SIZE];
  int c = 0;
  int *dev_a, *dev_c;
    
  hipMalloc((void **) &dev_a, SIZE*sizeof(int));
  hipMalloc((void **) &dev_c, sizeof(int));
  srand(time(0));
  for( i = 0 ; i < SIZE ; i++)
  {
    a[i] = (rand() % (1000 - 100 + 1)) + 100;
  }
  for( i = 0 ; i < SIZE ; i++)
  {
    printf("%d ",a[i]);
    if (i%10==0 && i!=0){
      printf("\n");
    }
  }
  hipMemcpy(dev_c , &c, sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(dev_a , a, SIZE*sizeof(int),hipMemcpyHostToDevice);
   // adjusting the number of threads per block
   dim3 threadsPerBlock(SIZE);
   dim3 blocksPerGrid(1, 1);
   if(SIZE > 1024){
           threadsPerBlock.x = 1024;
           blocksPerGrid.x = ceil(double(SIZE)/double(threadsPerBlock.x));
   }
  clock_t start = clock();
  sum<<<blocksPerGrid,threadsPerBlock>>>(dev_a,SIZE,dev_c);
  hipMemcpy(&c, dev_c, sizeof(int),hipMemcpyDeviceToHost);
  c = c / SIZE;
  clock_t end = clock();
  
  printf("avg =  %d ",c);
  printf("\nThe gpu took: %f milli-seconds.\n",(float)(end-start)/CLOCKS_PER_SEC);
    
  printf("\n");
  printf("avg =  %d ",c);
  hipFree(dev_a);
  hipFree(dev_c);
  return 0;
}
