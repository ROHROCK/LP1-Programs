
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#define SIZE 10000
// MAX_THREADS depends on type of GPU
#define MAX_THREADS 1024

__global__ void min(const int* __restrict__ input,int* minOut)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  atomicMin(minOut, input[i]);
}

int main()
{
  int i;
  int a[SIZE];
  int c;
  int *dev_a, *dev_c;
  hipMalloc((void **) &dev_a, SIZE*sizeof(int));
  hipMalloc((void **) &dev_c, sizeof(int));
  srand(time(0));
  for( i = 0 ; i < SIZE ; i++)
  {
    a[i] = (rand() % (1000 - 100 + 1)) + 100;
  }
  // a[0]= -10; just to check 
  hipMemcpy(dev_c , &c, sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(dev_a , a, SIZE*sizeof(int),hipMemcpyHostToDevice);
  // adjusting the number of threads per block
  dim3 threadsPerBlock(SIZE);
  dim3 blocksPerGrid(1, 1);
  if(SIZE > 1024){
          threadsPerBlock.x = 1024;
          blocksPerGrid.x = ceil(double(SIZE)/double(threadsPerBlock.x));
  }
  clock_t start = clock();
  min<<<blocksPerGrid,threadsPerBlock>>>(dev_a,dev_c);
  hipMemcpy(&c, dev_c, sizeof(int),hipMemcpyDeviceToHost);
  clock_t end = clock();
  printf("\nmin =  %d ",c);
  printf("\nThe time taken to execute is: %f",(float)(end-start)/CLOCKS_PER_SEC);
  hipFree(dev_a);
  hipFree(dev_c);
  return 0;
}
